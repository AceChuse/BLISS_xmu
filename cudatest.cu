    #include<hip/hip_runtime.h>    
    #include<iostream>  
    using namespace std;  
    const int nMax = 30000;  
    __global__ void addKernel(float *aaa,float *bbb, float *ccc)  
    {  
        //int i = blockIdx.x;  
        int i = threadIdx.x + blockIdx.x*blockDim.x;  
        ccc[i] = 0;  
        if (i < nMax)for (int j = 0; j < 500; j++)ccc[i] += aaa[i] * bbb[i];  
    }  
    void add(float *a, float *b,float *c,int i){  
        for (int j = 0; j<500; j++) c[i] += a[i] * b[i];  
    }  
    int main(){  
        float a[nMax], b[nMax], c[nMax];  
        float *devA, *devB, *devC;  
        clock_t startT, endT;  
        for (int i = 0; i < nMax; i++){  
            a[i] = i*1.010923;  
            b[i] = 2.13*i;  
        }  
        startT = clock();  
        hipMalloc((void**)&devA, nMax*sizeof(float));  
        hipMalloc((void**)&devB, nMax*sizeof(float));  
        hipMalloc((void**)&devC, nMax*sizeof(float));  
        endT = clock();  
        cout << "分配设备空间耗时 " << endT - startT << "ms"<<endl;  
      
      
        startT = clock();  
        hipMemcpy(devA, a,nMax*sizeof(float),hipMemcpyHostToDevice);  
        hipMemcpy(devB, b, nMax*sizeof(float), hipMemcpyHostToDevice);  
        endT = clock();  
        cout << "数据从主机写入设备耗时 " << endT - startT << "ms" << endl;  
      
        startT = clock();  
      
        hipEvent_t start1;  
        hipEventCreate(&start1);  
        hipEvent_t stop1;  
        hipEventCreate(&stop1);  
        hipEventRecord(start1, NULL);  
      
        addKernel<<<60,501>>>(devA, devB, devC);  
      
        hipEventRecord(stop1, NULL);  
        hipEventSynchronize(stop1);  
        float msecTotal1 = 0.0f;  
        hipEventElapsedTime(&msecTotal1, start1, stop1);  
        //cout << msecTotal1 << "ddd" << endl;  
        endT = clock();  
        cout << "GPU计算耗时 " << msecTotal1 << "ms" << endl;  
      
        startT = clock();  
        hipMemcpy(c, devC, nMax*sizeof(float), hipMemcpyDeviceToHost);  
        endT = clock();  
        cout << "数据从设备写入主机耗时 " << endT - startT << "ms" << endl;  
      
        cout <<"GPU计算结果 "<< c[nMax - 1] << endl;  
        for (int i = 0; i < nMax; i++){  
            a[i] = i*1.010923;  
            b[i] = 2.13*i;  
            c[i] = 0;  
        }  
        startT = clock();  
        for (int i = 0; i < nMax; i++){  
            add(a, b, c, i);  
        }  
        endT = clock();  
        cout << "CPU计算耗时 " << endT - startT << "ms" << endl;  
        cout << "CPU计算结果 " << c[nMax - 1] << endl;  
      
            //释放在设备上分配的空间  
        hipFree(devA);  
        hipFree(devB);  
        hipFree(devC);  
        cin >> a[0];  
        return 0;  
    }  
